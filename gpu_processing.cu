#include "hip/hip_runtime.h"
#include "header.h"

__global__ void PictureDevice_FILTER(png_byte *d_In, png_byte *d_Out, int height, int width, float *d_filt) {
    // Calculate thread coordinates
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;

    // Define shared memory for the filter
    __shared__ float shared_filt[25]; // 5x5 filter

    // Load filter into shared memory
    if (threadIdx.x < 5 && threadIdx.y < 5) {
        shared_filt[threadIdx.y * 5 + threadIdx.x] = d_filt[threadIdx.y * 5 + threadIdx.x];
    }
    __syncthreads();

    if (Row >= 2 && Row < height - 2 && Col >= 2 && Col < width - 2) {
        float out;
        png_byte b;

        // Loop over the three color channels
        for (int color = 0; color < 3; color++) {
            out = 0.0;
            // Loop over the filter window
            for (int i = -2; i <= 2; i++) {
                for (int j = -2; j <= 2; j++) {

                    // Ensure coalesced access by calculating address once and reusing
                    int img_idx = ((Row + i) * width + (Col + j)) * 3 + color;
                    out += shared_filt[(i+2) * 5 + (j+2)] * d_In[img_idx];
                }
            }
            // Clamp the result to the range [0, 255]
            b = (png_byte)fminf(fmaxf(out, 0.0), 255.0);
            d_Out[(Row * width + Col) * 3 + color] = b;
        }
    }
}

void execute_jobs_gpu(PROCESSING_JOB **jobs) {
    int count = 0;
    float *d_sharpen_filter = nullptr, *d_box_blur_filter = nullptr, *d_edge_filter = nullptr;
    png_byte *d_In = nullptr, *d_Out = nullptr;
    size_t maxNumPixels = 0;

    // Determine the maximum number of pixels to allocate memory once
    while (jobs[count] != nullptr) {
        size_t numPixels = jobs[count]->height * jobs[count]->width * 3; // 3 for RGB channels
        if (numPixels > maxNumPixels) {
            maxNumPixels = numPixels;
        }
        count++;
    }

    // Allocate memory on the device
    hipMalloc((void **)&d_In, maxNumPixels * sizeof(png_byte));
    hipMalloc((void **)&d_Out, maxNumPixels * sizeof(png_byte));
    hipMalloc((void **)&d_sharpen_filter, 25 * sizeof(float));
    hipMalloc((void **)&d_box_blur_filter, 25 * sizeof(float));
    hipMalloc((void **)&d_edge_filter, 25 * sizeof(float));

    // Copy filters to device
    hipMemcpy(d_sharpen_filter, sharpen_filter, 25 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_box_blur_filter, box_blur_filter, 25 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_edge_filter, edge_detect_filter, 25 * sizeof(float), hipMemcpyHostToDevice);

    // Initial creation of data and copying of data for the first image job
    char* filename = jobs[0]->source_name;
    size_t numPixels = jobs[0]->height * jobs[0]->width * 3; // 3 for RGB channels
    hipMemcpy(d_In, jobs[0]->source_raw, numPixels * sizeof(png_byte), hipMemcpyHostToDevice);
    dim3 blocks((jobs[0]->width - 4 + 15) / 16, (jobs[0]->height - 4 + 15) / 16);
    dim3 threads(16, 16);

    float *d_filter;
    switch (jobs[0]->processing_algo) {
        case SHARPEN:
            d_filter = d_sharpen_filter;
            break;
        case BLUR:
            d_filter = d_box_blur_filter;
            break;
        case EDGE:
            d_filter = d_edge_filter;
            break;
        default:
            return; // Invalid processing algorithm
    }

    // Run the kernel for the first image
    PictureDevice_FILTER<<<blocks, threads>>>(d_In, d_Out, jobs[0]->height, jobs[0]->width, d_filter);

    // Copy result back to host
    hipMemcpy(jobs[0]->dest_raw, d_Out, numPixels * sizeof(png_byte), hipMemcpyDeviceToHost);

    // Start loop at 1 since we already processed the first image
    count = 1;
    while (jobs[count] != nullptr) {

        char* current_filename = jobs[count]->source_name;

        // If the filename is different, we need to copy the new image to the device
        if (strcmp(filename, current_filename) != 0) {
            filename = current_filename;
            numPixels = jobs[count]->height * jobs[count]->width * 3; // 3 for RGB channels
            hipMemcpy(d_In, jobs[count]->source_raw, numPixels * sizeof(png_byte), hipMemcpyHostToDevice);
            blocks = dim3((jobs[count]->width - 4 + 15) / 16, (jobs[count]->height - 4 + 15) / 16);
        }

        // Determine filter
        switch (jobs[count]->processing_algo) {
            case SHARPEN:
                d_filter = d_sharpen_filter;
                break;
            case BLUR:
                d_filter = d_box_blur_filter;
                break;
            case EDGE:
                d_filter = d_edge_filter;
                break;
            default:
                return; // Invalid processing algorithm
        }

        // Launch the kernel
        PictureDevice_FILTER<<<blocks, threads>>>(d_In, d_Out, jobs[count]->height, jobs[count]->width, d_filter);

        // Copy result back to host
        hipMemcpy(jobs[count]->dest_raw, d_Out, numPixels * sizeof(png_byte), hipMemcpyDeviceToHost);

        count++;
    }

    // Free device memory
    hipFree(d_In);
    hipFree(d_Out);
    hipFree(d_filter);
}
