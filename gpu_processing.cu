#include "hip/hip_runtime.h"
#include "header.h"

__global__ void PictureDevice_FILTER(png_byte *d_In, png_byte *d_Out, int height, int width, float *d_filt) {
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;

    if (Row >= 2 && Row < height - 2 && Col >= 2 && Col < width - 2) {
        float out;
        png_byte b;

        for (int color = 0; color < 3; color++) {
            out = 0.0;
            for (int i = -2; i <= 2; i++) {
                for (int j = -2; j <= 2; j++) {
                    out += d_filt[(i+2) * 5 + (j+2)] * d_In[((Row + i) * width + (Col + j)) * 3 + color];
                }
            }
            b = (png_byte)fminf(fmaxf(out, 0.0), 255.0);
            d_Out[(Row * width + Col) * 3 + color] = b;
        }
    }
}

void execute_jobs_gpu(PROCESSING_JOB **jobs) {
    int count = 0;
    float *d_filter, *h_filter;
    png_byte *d_In, *d_Out;
    size_t numPixels;

    while (jobs[count] != NULL) {
        numPixels = jobs[count]->height * jobs[count]->width * 3; // 3 for RGB channels

        // Allocate memory for filter on the host
        h_filter = getAlgoFilterByType(jobs[count]->processing_algo);

        // Allocate memory on the device
        hipMalloc((void **)&d_In, numPixels * sizeof(png_byte));
        hipMalloc((void **)&d_Out, numPixels * sizeof(png_byte));
        hipMalloc((void **)&d_filter, 25 * sizeof(float)); // Assuming a 5x5 filter

        // Copy data from host to device
        hipMemcpy(d_In, jobs[count]->source_raw, numPixels * sizeof(png_byte), hipMemcpyHostToDevice);
        hipMemcpy(d_filter, h_filter, 25 * sizeof(float), hipMemcpyHostToDevice);

        // Set up the execution configuration
        dim3 blocks((jobs[count]->width - 4 + 15) / 16, (jobs[count]->height - 4 + 15) / 16);
        dim3 threads(16, 16);

        // Launch the kernel
        PictureDevice_FILTER<<<blocks, threads>>>(d_In, d_Out, jobs[count]->height, jobs[count]->width, d_filter);

        // Copy result back to host
        hipMemcpy(jobs[count]->dest_raw, d_Out, numPixels * sizeof(png_byte), hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(d_In);
        hipFree(d_Out);
        hipFree(d_filter);

        count++;
    }
}
