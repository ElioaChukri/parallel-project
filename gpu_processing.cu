#include "hip/hip_runtime.h"
#include "header.h"

__global__ void PictureDevice_FILTER(png_byte *d_In, png_byte *d_Out, int height, int width, float *d_filt) {
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;

    if (Row >= 2 && Row < height - 2 && Col >= 2 && Col < width - 2) {
        float out;
        png_byte b;

        for (int color = 0; color < 3; color++) {
            out = 0.0;
            for (int i = -2; i <= 2; i++) {
                for (int j = -2; j <= 2; j++) {
                    out += d_filt[(i+2) * 5 + (j+2)] * d_In[((Row + i) * width + (Col + j)) * 3 + color];
                }
            }
            b = (png_byte)fminf(fmaxf(out, 0.0), 255.0);
            d_Out[(Row * width + Col) * 3 + color] = b;
        }
    }
}

void execute_jobs_gpu(PROCESSING_JOB **jobs) {
    int count = 0;
    float *d_filter = nullptr;
    png_byte *d_In = nullptr, *d_Out = nullptr;
    size_t maxNumPixels = 0;

    // Determine the maximum number of pixels to allocate memory once
    while (jobs[count] != NULL) {
        size_t numPixels = jobs[count]->height * jobs[count]->width * 3; // 3 for RGB channels
        if (numPixels > maxNumPixels) {
            maxNumPixels = numPixels;
        }
        count++;
    }

    // Allocate memory on the device
    hipMalloc((void **)&d_In, maxNumPixels * sizeof(png_byte));
    hipMalloc((void **)&d_Out, maxNumPixels * sizeof(png_byte));
    hipMalloc((void **)&d_filter, 25 * sizeof(float)); // Assuming a 5x5 filter

    count = 0;
    while (jobs[count] != NULL) {
        size_t numPixels = jobs[count]->height * jobs[count]->width * 3; // 3 for RGB channels
        float *h_filter = getAlgoFilterByType(jobs[count]->processing_algo);

        // Copy data from host to device
        hipMemcpy(d_In, jobs[count]->source_raw, numPixels * sizeof(png_byte), hipMemcpyHostToDevice);
        hipMemcpy(d_filter, h_filter, 25 * sizeof(float), hipMemcpyHostToDevice);

        // Set up the execution configuration
        dim3 blocks((jobs[count]->width - 4 + 15) / 16, (jobs[count]->height - 4 + 15) / 16);
        dim3 threads(16, 16);

        // Launch the kernel
        PictureDevice_FILTER<<<blocks, threads>>>(d_In, d_Out, jobs[count]->height, jobs[count]->width, d_filter);

        // Copy result back to host
        hipMemcpy(jobs[count]->dest_raw, d_Out, numPixels * sizeof(png_byte), hipMemcpyDeviceToHost);

        count++;
    }

    // Free device memory
    hipFree(d_In);
    hipFree(d_Out);
    hipFree(d_filter);
}
